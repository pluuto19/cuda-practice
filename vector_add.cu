
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#define N 512

void host_add(int *a, int *b, int *c) {
    for(int idx=0;idx<N;idx++)
        c[idx] = a[idx] + b[idx];
}

void fill_array(int *data) {
    for(int idx=0;idx<N;idx++)
        data[idx] = idx;
}
void print_output(int *a, int *b, int*c) {
    for(int idx=0;idx<N;idx++)
        printf("\n %d + %d = %d", a[idx] , b[idx], c[idx]);
}
__global__ void device_add(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}
int main(void) {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);
    fill_array(a);
    fill_array(b);

    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N* sizeof(int), hipMemcpyHostToDevice);

    device_add<<<1,1>>>(d_a,d_b,d_c);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    print_output(a,b,c);

    free(a); free(b); free(c);

    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}

