
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void helloGPU(void){
    printf("Hello from GPU thread %d, %d", threadIdx.x, blockIdx.x);
}

int main(void){
    printf("This is host");
    helloGPU<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}