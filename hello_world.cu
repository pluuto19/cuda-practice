
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void helloGPU(void){
    printf("Hello from GPU thread %d, %d\n", threadIdx.x, blockIdx.x);
}

int main(void){
    printf("This is host\n");
    helloGPU<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}