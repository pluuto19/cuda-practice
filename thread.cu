
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void mythreadfunc(void){
    printf("Thread and Block ID %d,%d,%d,%d \n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
}

int main(void){
    printf("1 thread and 1 block\n");
    mythreadfunc<<<1,1>>>();
    hipDeviceSynchronize();
    printf("10 threads in one block\n");
    mythreadfunc<<<1,10>>>();
    hipDeviceSynchronize();
    printf("10 blocks and 1 thread\n");
    mythreadfunc<<<10,1>>>();
    hipDeviceSynchronize();
    printf("10 blocks and 10 threads per block\n");
    mythreadfunc<<<10,10>>>();
    hipDeviceSynchronize();
}